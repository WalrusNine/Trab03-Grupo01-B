/*
**	Disciplina:	SSC-0143 Programacao Concorrente
**
**	Docente:	Prof. Julio Cezar Estrella
**
**	Discentes:	Andre Miguel Coelho Leite	8626249
**			Laerte Vidal Junior		7557800
**
**	Trabalho 3:	Smooth com CUDA
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define RGB 0
#define RBG 1
#define GRB 2
#define GBR 3
#define BRG 4 
#define BGR 5

/*
**	Image is stored in an array of PIXELs,
**	which have red, green and blue values
**	as unsigned chars (RGB) or just one
**	unsigned char(GRS).
**
**
**	SMOOTH
**	5x5
**	(i-2,j-2)	(i-1,j-2)	(i  ,j-2)	(i+1,j-2)	(i+2,j-2)
**	(i-2,j-1)	(i-1,j-1)	(i  ,j-1)	(i+1,j-1)	(i+2,j-1)
**	(i-2,  j)	(i-1,  j)	(i  ,  j)	(i+1,  j)	(i+2,  j)
**	(i-2,j+1)	(i-1,j+1)	(i  ,j+1)	(i+1,j+1)	(i+2,j+1)
**	(i-2,j+2)	(i-1,j+2)	(i  ,j+2)	(i+1,j+2)	(i+2,j+2)
*/

/* RGB PIXEL */
typedef struct{
	unsigned char r, g, b;
}PIXELRGB;

/* GRAYSCALE PIXEL */
typedef struct{
	unsigned char i;
}PIXELGRS;

/* UNION for keeping either grayscale or rgb PIXEL */
typedef union{
	PIXELGRS grs;
	PIXELRGB rgb;
} PIXEL;

typedef struct image{
	PIXEL* pixel;
	int width, height;
}IMAGE;

/* Prototypes */
IMAGE* read_ppm_image();
void write_ppm(const char *,IMAGE*,int);
void delete_image(IMAGE**);

__global__ void smooth_grs(PIXEL*,PIXEL*, int, int);
__global__ void smooth_rgb(PIXEL*,PIXEL*, int, int);

int timeval_subtract(struct timeval*, struct timeval*, struct timeval*);

/* Globals */
int grayscale = 0;

int main(int argc, char** argv)
{
	IMAGE* 	image;
	PIXEL*	gpixels;

	/* Time variables */
	struct timeval t_begin, t_end, t_diff;

	/* Read image */	
	image = read_ppm_image();
	gpixels = image->pixel;

	/* Get time start */
	gettimeofday(&t_begin, NULL);

	/* Size */
	size_t size = (image->width)*(image->height)*sizeof(PIXEL);

	/* Device arrays */
	PIXEL* d_pixels_in;
	hipMalloc(&d_pixels_in, size);
	hipMemcpy(d_pixels_in, gpixels, size, hipMemcpyHostToDevice);

	PIXEL* d_pixels_out;
	hipMalloc(&d_pixels_out, size);

	/* Setup blocks and threads */
	dim3 threadsPerBlock( 32, 32 );
	dim3 numBlocks( (image->width) / threadsPerBlock.x, (image->height) / threadsPerBlock.y );
	
	/* Run smooth */
	if (grayscale) smooth_grs<<<numBlocks, threadsPerBlock>>>(d_pixels_in, d_pixels_out, image->width, image->height);
	else smooth_rgb<<<numBlocks, threadsPerBlock>>>(d_pixels_in, d_pixels_out, image->width, image->height);

	/* Get time end */
	gettimeofday(&t_end, NULL);

	/* Get diff time and print in stderr */
	timeval_subtract(&t_diff, &t_end, &t_begin);
	fprintf(stderr, "%ld.%06ld\n", t_diff.tv_sec, t_diff.tv_usec);

	/* Copy results */
	hipMemcpy(image->pixel, d_pixels_out, size, hipMemcpyDeviceToHost);

	/* Write resulting image */
	write_ppm("out.ppm",image,RGB);

	/* Free memory */
	delete_image(&image);

	hipFree(d_pixels_in);
	hipFree(d_pixels_out);
	
	return EXIT_SUCCESS;
}

	
IMAGE* read_ppm_image()
{
	/*
	**	Snippet adapted from:
	**	http://stackoverflow.com/questions/2693631/read-ppm-file-and-store-it-in-an-array-coded-with-c
	*/
	
	FILE* fp = fopen("in.ppm", "rb");
	
	/*
	**	PX
	**	n_columns m_rows
	**	max_color
	**	row_1 -> column_1  ...  column_n
	**	...
	**	row_n -> column_1  ...  column_n
	**
	**	each row is in format (255 255 255) => (R G B)
	*/

	/* Get Type PPM */
	char type[3];
	fscanf(fp, "%s\n", type);
	if (type[1] == '5') grayscale = 1;
	
	/* Check for comments */
	char c = getc(fp);
	while (c == '#') {
		while (getc(fp) != '\n');
		c = getc(fp);
	}
	ungetc(c, fp);

	/* Get Size */
	int width, height;
	fscanf(fp, "%d %d\n", &width, &height);

	/* Get Max Color */
	fscanf(fp, "%*d\n");

	/* Create Image */
	IMAGE* image	= (IMAGE*)malloc(sizeof(IMAGE));
	image->width 	= width;
	image->height 	= height;
	image->pixel 	= (PIXEL *)malloc(width*height*sizeof(PIXEL));

	/* Read image's pixel data */
	if (grayscale){
		int i, j;
		for (i = 0; i < height; ++i){
			for (j = 0; j < width; ++j){
				fread(&(image->pixel[i*width + j].grs.i), 1, 1, fp);
			}
		}
	}
	else fread(image->pixel, sizeof(PIXEL),width*height, fp);

	fclose(fp);

	return image;
}

__global__ void smooth_rgb(PIXEL* in, PIXEL* out, int width, int height){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if( i >= width || j >= height )
		return;

	int k, l;
	int red, green, blue;
	red = green = blue = 0;
	for (k = -2; k <= 2; k++){
		for (l = -2; l <= 2; l++){
			if (i+k >= 0 && i+k < width && j+l >= 0 && j+l < height){
				red 	+= in[(j+l)*width + (i+k)].rgb.r;
				green 	+= in[(j+l)*width + (i+k)].rgb.g;
				blue 	+= in[(j+l)*width + (i+k)].rgb.b;
			}
		}
	}
	out[(j)*width + (i)].rgb.r = (red / 25);
	out[(j)*width + (i)].rgb.g = (green / 25);
	out[(j)*width + (i)].rgb.b = (blue / 25);
}

__global__ void smooth_grs(PIXEL* in, PIXEL* out, int width, int height){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if( i >= width || j >= height )
		return;

	int k, l;
	int mean = 0;
	for (k = -2; k <= 2; k++){
		for (l = -2; l <= 2; l++){
			if (i+k >= 0 && i+k < width && j+l >= 0 && j+l < height){
				mean += in[(j+l)*width + (i+k)].grs.i;
			}
		}
	}
	out[(j)*width + (i)].grs.i = (mean / 25);
}

void write_ppm(const char *fname,IMAGE* image,int m){
	FILE *fp = fopen(fname, "wb");
	
	/* Put header */
	if (grayscale) fprintf(fp, "P5\n");
	else fprintf(fp, "P6\n");

	/* Put size */
	fprintf(fp, "%d %d\n",image->width, image->height);
	int width = image->width, height = image->height;

	/* RGB component depth */
	fprintf(fp, "%d\n", 255);

	/* Pixel data */
	int i, j;
	for (j = 0; j < height; j++){
		for (i = 0; i < width; i++){
			if (grayscale){
				fwrite(&(((image->pixel)[j*width + i]).grs.i), sizeof(unsigned char),1, fp);
			} else{
				fwrite(&(image->pixel[j*width + i].rgb.r), sizeof(unsigned char),1, fp);
				fwrite(&(image->pixel[j*width + i].rgb.g), sizeof(unsigned char),1, fp);
				fwrite(&(image->pixel[j*width + i].rgb.b), sizeof(unsigned char),1, fp);
			}
		}
	}
	
	fclose(fp);
}

void delete_image(IMAGE** image){
	if (*image != NULL){
		free((*image)->pixel);
		free(*image);
		*image = NULL;
	}
}


/*
**	Snippet from:
**	http://stackoverflow.com/questions/1468596/calculating-elapsed-time-in-a-c-program-in-milliseconds
*/

/* Return 1 if the difference is negative, otherwise 0.  */
int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
	long int diff = (t2->tv_usec + 1000000 * t2->tv_sec) - (t1->tv_usec + 1000000 * t1->tv_sec);
	result->tv_sec = diff / 1000000;
	result->tv_usec = diff % 1000000;

	return (diff<0);
}
